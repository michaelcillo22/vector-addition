#include <iostream>
#include <hip/hip_runtime.h>
#include <chrono>

#define N 1000000  // Vector size

using namespace std;
using namespace std::chrono;

// CUDA kernel for vector addition
__global__ void vectorAdd(float *A, float *B, float *C, int n) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < n) {
        C[i] = A[i] + B[i];
    }
}

// Function to run vector addition on GPU
void gpuVectorAdd(float *A, float *B, float *C, int n) {
    float *d_A, *d_B, *d_C;
    size_t size = n * sizeof(float);

    // Allocate memory on GPU
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    // Copy data from CPU to GPU
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Define grid and block size
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    // Start timing
    auto start = high_resolution_clock::now();

    // Launch kernel
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, n);

    // Synchronize to ensure completion
    hipDeviceSynchronize();

    // End timing
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "GPU Execution Time: " << duration.count() << " µs" << endl;

    // Copy result back to CPU
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

// CPU Implementation
void cpuVectorAdd(float *A, float *B, float *C, int n) {
    auto start = high_resolution_clock::now();
    
    for (int i = 0; i < n; i++) {
        C[i] = A[i] + B[i];
    }
    
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start);
    cout << "CPU Execution Time: " << duration.count() << " µs" << endl;
}

// Main function
int main() {
    float *A, *B, *C_cpu, *C_gpu;
    A = new float[N];
    B = new float[N];
    C_cpu = new float[N];
    C_gpu = new float[N];

    // Initialize arrays with random values
    for (int i = 0; i < N; i++) {
        A[i] = static_cast<float>(rand()) / RAND_MAX;
        B[i] = static_cast<float>(rand()) / RAND_MAX;
    }

    // Run CPU addition
    cpuVectorAdd(A, B, C_cpu, N);

    // Run GPU addition
    gpuVectorAdd(A, B, C_gpu, N);

    // Verify correctness
    bool correct = true;
    for (int i = 0; i < N; i++) {
        if (abs(C_cpu[i] - C_gpu[i]) > 1e-5) {
            correct = false;
            break;
        }
    }

    if (correct) {
        cout << "Results are CORRECT!" << endl;
    } else {
        cout << "Results MISMATCH!" << endl;
    }

    // Cleanup
    delete[] A;
    delete[] B;
    delete[] C_cpu;
    delete[] C_gpu;

    return 0;
}
